#include "hip/hip_runtime.h"
# include "adjoint_state.cuh"

void Adjoint_State::set_specifications()
{
    inversion_name = "adjoint_state_";
    inversion_method = "Adjoint-State First-Arrival Tomography";

    nSweeps = 4;
    meshDim = 2;
    nThreads = 32;      

    cell_area = modeling->dx*modeling->dz;

    total_levels = modeling->nxx + modeling->nzz - 1;

    m = new float[modeling->nPoints]();
    v = new float[modeling->nPoints]();

    source_grad = new float[modeling->matsize]();
    source_comp = new float[modeling->matsize]();
    
    adjoint_grad = new float[modeling->matsize]();
    adjoint_comp = new float[modeling->matsize]();

    gradient = new float[modeling->nPoints]();
    illumination = new float[modeling->nPoints]();     

    hipMalloc((void**)&(d_T), modeling->matsize*sizeof(float));

    hipMalloc((void**)&(d_source_grad), modeling->matsize*sizeof(float));
    hipMalloc((void**)&(d_source_comp), modeling->matsize*sizeof(float));

    hipMalloc((void**)&(d_adjoint_grad), modeling->matsize*sizeof(float));
    hipMalloc((void**)&(d_adjoint_comp), modeling->matsize*sizeof(float));
}

void Adjoint_State::apply_inversion_technique()
{
    initialization();

    hipMemcpy(d_T, modeling->T, modeling->matsize*sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_source_grad, source_grad, modeling->matsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_source_comp, source_comp, modeling->matsize*sizeof(float), hipMemcpyHostToDevice);    

    hipMemcpy(d_adjoint_grad, adjoint_grad, modeling->matsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_adjoint_comp, adjoint_comp, modeling->matsize*sizeof(float), hipMemcpyHostToDevice);    

    int min_level = std::min(modeling->nxx, modeling->nzz);
    int max_level = std::max(modeling->nxx, modeling->nzz);

    int z_offset, x_offset, n_elements;

    for (int sweep = 0; sweep < nSweeps; sweep++)
    { 
        int zd = (sweep == 2 || sweep == 3) ? -1 : 1; 
        int xd = (sweep == 0 || sweep == 2) ? -1 : 1;

        for (int level = 0; level < total_levels; level++)
        {
            z_offset = (sweep == 0) ? ((level < modeling->nxx) ? 0 : level - modeling->nxx + 1) :
                       (sweep == 1) ? ((level < modeling->nzz) ? modeling->nzz - level - 1 : 0) :
                       (sweep == 2) ? ((level < modeling->nzz) ? level : modeling->nzz - 1) :
                                      ((level < modeling->nxx) ? modeling->nzz - 1 : modeling->nzz - 1 - (level - modeling->nxx + 1));

            x_offset = (sweep == 0) ? ((level < modeling->nxx) ? level : modeling->nxx - 1) :
                       (sweep == 1) ? ((level < modeling->nzz) ? 0 : level - modeling->nzz + 1) :
                       (sweep == 2) ? ((level < modeling->nzz) ? modeling->nxx - 1 : modeling->nxx - 1 - (level - modeling->nzz + 1)) :
                                      ((level < modeling->nxx) ? modeling->nxx - level - 1 : 0);

            n_elements = (level < min_level) ? level + 1 : 
                         (level >= max_level) ? total_levels - level : 
                         total_levels - min_level - max_level + level;

            nBlocks = (int)((n_elements + nThreads - 1) / nThreads);

            inner_sweep<<<nBlocks, nThreads>>>(d_T, d_adjoint_grad, d_adjoint_comp, d_source_grad, d_source_comp, x_offset, z_offset, xd, zd, modeling->nxx, modeling->nzz, modeling->dx, modeling->dz);

            hipDeviceSynchronize();    
        }
    }

    hipMemcpy(adjoint_grad, d_adjoint_grad, modeling->matsize*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(adjoint_comp, d_adjoint_comp, modeling->matsize*sizeof(float), hipMemcpyDeviceToHost);

    # pragma omp parallel for reduction(+:gradient[:modeling->nPoints])
    for (int index = 0; index < modeling->nPoints; index++) 
    {
        int i = (int) (index % modeling->nz);    
        int j = (int) (index / modeling->nz);  

        int indp = i + j*modeling->nz; 
        int indb = (i + modeling->nb) + (j + modeling->nb)*modeling->nzz;

        gradient[indp] += (adjoint_grad[indb] / (adjoint_comp[indb] + 1e-6f))*cell_area / modeling->geometry->nrel;
    }
}

void Adjoint_State::initialization()
{   
    # pragma omp parallel for
    for (int index = 0; index < modeling->matsize; index++) 
    {
        source_grad[index] = 0.0f;    
        source_comp[index] = 0.0f;    
        
        adjoint_grad[index] = 1e6f;
        adjoint_comp[index] = 1e6f;

        int i = (int) (index % modeling->nzz);    
        int j = (int) (index / modeling->nzz);  

        if ((i == 0) || (i == modeling->nzz - 1) || 
            (j == 0) || (j == modeling->nxx - 1)) 
        {    
            adjoint_grad[index] = 0.0f;        
            adjoint_comp[index] = 0.0f;        
        }
    }

    int skipped = modeling->srcId * modeling->geometry->spread[modeling->srcId];

    int sIdx = (int)(modeling->geometry->xsrc[modeling->geometry->sInd[modeling->srcId]] / modeling->dx) + modeling->nb;
    int sIdz = (int)(modeling->geometry->zsrc[modeling->geometry->sInd[modeling->srcId]] / modeling->dz) + modeling->nb;

    float So = modeling->S[sIdz + sIdx*modeling->nzz];

    int spread = 0;

    for (modeling->recId = modeling->geometry->iRec[modeling->srcId]; modeling->recId < modeling->geometry->fRec[modeling->srcId]; modeling->recId++)
    {
        int rIdx = (int)(modeling->geometry->xrec[modeling->recId] / modeling->dx) + modeling->nb;
        int rIdz = (int)(modeling->geometry->zrec[modeling->recId] / modeling->dz) + modeling->nb;

        for (int i = 0; i < 3; i++)
        {
            for (int j = 0; j < 3; j++)
            {
                int xi = rIdx + (j - 1);
                int zi = rIdz + (i - 1);

                float X = sqrtf(powf((sIdx - xi)*modeling->dx, 2.0f) + powf((sIdz - zi)*modeling->dz, 2.0f));

                int index = zi + xi*modeling->nzz;
                
                source_grad[index] += (dobs[spread + skipped] - modeling->T[index]) / cell_area;
                source_comp[index] += 1.0f / (X*X*So);
            }
        }

        ++spread;
    }   
} 

void Adjoint_State::optimization()
{   
    float gdot = 0.0f;
    #pragma omp parallel for reduction(+:gdot)
    for (int index = 0; index < modeling->nPoints; index++)
        gdot += gradient[index]*gradient[index];
    
    float beta1 = 0.5f;
    float beta2 = 0.9f;

    float epsilon = 1e-8f;

    for (int index = 0; index < modeling->nPoints; index++)
    {
        gradient[index] *= 1.0f / gdot;

        m[index] = beta1*m[index] + (1.0f - beta1)*gradient[index];
        
        v[index] = beta2*v[index] + (1.0f - beta2)*gradient[index]*gradient[index];

        float m_hat = m[index] / (1.0f - powf(beta1, iteration));
        
        float v_hat = v[index] / (1.0f - powf(beta2, iteration));

        perturbation[index] = max_slowness_variation*m_hat / (sqrtf(v_hat) + epsilon);
    }

    memset(gradient, 0.0f, modeling->nPoints);
}

__global__ void inner_sweep(float * T, float * adjoint_grad, float * adjoint_comp, float * source_grad, float * source_comp, int x_offset, int z_offset, int xd, int zd, int nxx, int nzz, float dx, float dz)
{
    int element = blockIdx.x*blockDim.x + threadIdx.x;

    int i = z_offset + zd*element;
    int j = x_offset + xd*element;    

    if ((i > 0) && (i < nzz-1) && (j > 0) && (j < nxx-1))
    {
        float a1 = -1.0f*(T[i + j*nzz] - T[i + (j-1)*nzz]) / dx;
        float ap1 = 0.5f*(a1 + fabsf(a1));
        float am1 = 0.5f*(a1 - fabsf(a1));

        float a2 = -1.0f*(T[i + (j+1)*nzz] - T[i + j*nzz]) / dx;
        float ap2 = 0.5f*(a2 + fabsf(a2));
        float am2 = 0.5f*(a2 - fabsf(a2));

        float c1 = -1.0f*(T[i + j*nzz] - T[(i-1) + j*nzz]) / dz;
        float cp1 = 0.5f*(c1 + fabsf(c1));
        float cm1 = 0.5f*(c1 - fabsf(c1));

        float c2 = -1.0f*(T[(i+1) + j*nzz] - T[i + j*nzz]) / dz;
        float cp2 = 0.5f*(c2 + fabsf(c2));
        float cm2 = 0.5f*(c2 - fabsf(c2));

        float d = (ap2 - am1) / dx + (cp2 - cm1) / dz;

        if (fabsf(d) < 1e-6f)
        {
            adjoint_grad[i + j*nzz] = 0.0f;    
            adjoint_comp[i + j*nzz] = 0.0f;    
        }
        else
        {
            float eg = (ap1*adjoint_grad[i + (j-1)*nzz] - am2*adjoint_grad[i + (j+1)*nzz]) / dx +
                       (cp1*adjoint_grad[(i-1) + j*nzz] - cm2*adjoint_grad[(i+1) + j*nzz]) / dz;

            float ec = (ap1*adjoint_comp[i + (j-1)*nzz] - am2*adjoint_comp[i + (j+1)*nzz]) / dx +
                       (cp1*adjoint_comp[(i-1) + j*nzz] - cm2*adjoint_comp[(i+1) + j*nzz]) / dz;

            float fg = (eg + source_grad[i + j*nzz]) / d;
            float fc = (ec + source_comp[i + j*nzz]) / d;

            if (adjoint_grad[i + j*nzz] > fg) adjoint_grad[i + j*nzz] = fg;
            if (adjoint_comp[i + j*nzz] > fc) adjoint_comp[i + j*nzz] = fc;
        }
    }
}

