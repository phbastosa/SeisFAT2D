#include "hip/hip_runtime.h"
# include "kirchhoff.cuh"

void Kirchhoff::set_specifications()
{
    hipMalloc((void**)&(d_Tr), modeling->nPoints*sizeof(float));
    hipMalloc((void**)&(d_Ts), modeling->nPoints*sizeof(float));
    hipMalloc((void**)&(d_image), modeling->nPoints*sizeof(float));

    hipMalloc((void**)&(d_seismic), modeling->nt*modeling->max_spread*sizeof(float));

    nThreads = 256;
    nBlocks = (int)((modeling->nPoints + nThreads - 1) / nThreads);
}

void Kirchhoff::run_cross_correlation()
{
    hipMemset(d_image, 0.0f, modeling->nPoints*sizeof(float));

    for (modeling->srcId = 0; modeling->srcId < modeling->geometry->nrel; modeling->srcId++)
    {
        read_seismic_data();

        modeling->show_information();

        std::cout << "\nKirchhoff depth migration: computing image matrix\n\n";

        modeling->initialization();
        modeling->forward_solver();

        modeling->reduce_boundary(modeling->T, Ts);

        hipMemcpy(d_Ts, Ts, modeling->nPoints*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_seismic, seismic, modeling->nt*modeling->geometry->spread[modeling->srcId]*sizeof(float), hipMemcpyHostToDevice);

        int spread = 0;

        for (modeling->recId = modeling->geometry->iRec[modeling->srcId]; modeling->recId < modeling->geometry->fRec[modeling->srcId]; modeling->recId++)
        {
            import_binary_float(output_table_folder + "traveltimes_receiver_" + std::to_string(modeling->recId+1) + ".bin", Tr, modeling->nPoints);
            
            hipMemcpy(d_Tr, Tr, modeling->nPoints*sizeof(float), hipMemcpyHostToDevice);

            cross_correlation<<<nBlocks, nThreads>>>(d_seismic, d_Ts, d_Tr, d_image, modeling->nPoints, spread, modeling->nt, modeling->dt);

            ++spread;
        }
    }

    hipMemcpy(image, d_image, modeling->nPoints*sizeof(float), hipMemcpyDeviceToHost);
}

__global__ void cross_correlation(float * seismic, float * Ts, float * Tr, float * image, int nPoints, int spread, int nt, float dt)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < nPoints)
    {
        float T = Ts[index] + Tr[index]; 
    
        int tId = (int)(T / dt);

        if (tId < nt) image[index] += seismic[tId + spread*nt];
    }
}

