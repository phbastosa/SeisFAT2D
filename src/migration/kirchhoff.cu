#include "hip/hip_runtime.h"
# include "kirchhoff.cuh"

void Kirchhoff::set_specifications()
{
    hipMalloc((void**)&(d_Tr), modeling->nPoints*sizeof(float));
    hipMalloc((void**)&(d_Ts), modeling->nPoints*sizeof(float));
    hipMalloc((void**)&(d_image), modeling->nPoints*sizeof(float));
    hipMalloc((void**)&(d_gather), modeling->nz*modeling->max_spread*sizeof(float));
    hipMalloc((void**)&(d_seismic), modeling->nt*modeling->max_spread*sizeof(float));

    nThreads = 256;
    nBlocks = (int)((modeling->nPoints + nThreads - 1) / nThreads);
}

void Kirchhoff::run_cross_correlation()
{
    hipMemset(d_image, 0.0f, modeling->nPoints*sizeof(float));

    for (modeling->srcId = 0; modeling->srcId < modeling->geometry->nrel; modeling->srcId++)
    {
        read_seismic_data();

        modeling->show_information();

        std::cout << "\nKirchhoff depth migration: computing image matrix\n";

        modeling->initialization();
        modeling->forward_solver();

        modeling->reduce_boundary(modeling->T, Ts);

        hipMemcpy(d_Ts, Ts, modeling->nPoints*sizeof(float), hipMemcpyHostToDevice);
        
        hipMemset(d_gather, 0.0f, modeling->nz*modeling->geometry->spread[modeling->srcId]*sizeof(float));
        hipMemcpy(d_seismic, seismic, modeling->nt*modeling->geometry->spread[modeling->srcId]*sizeof(float), hipMemcpyHostToDevice);
        
        int spread = 0;

        float sx = modeling->geometry->xsrc[modeling->geometry->sInd[modeling->srcId]];

        for (modeling->recId = modeling->geometry->iRec[modeling->srcId]; modeling->recId < modeling->geometry->fRec[modeling->srcId]; modeling->recId++)
        {
            import_binary_float(output_table_folder + "traveltimes_receiver_" + std::to_string(modeling->recId+1) + ".bin", Tr, modeling->nPoints);

            float rx = modeling->geometry->xrec[modeling->recId];

            float cmp = sx + 0.5f*(rx - sx);

            hipMemcpy(d_Tr, Tr, modeling->nPoints*sizeof(float), hipMemcpyHostToDevice);

            cross_correlation<<<nBlocks, nThreads>>>(d_Ts, d_Tr, d_image, d_gather, d_seismic, aperture, cmp, modeling->nPoints, spread, modeling->nz, modeling->nt, modeling->dt, modeling->dx, modeling->dz);

            ++spread;
        }

        hipMemcpy(gather, d_gather, modeling->nz*modeling->geometry->spread[modeling->srcId]*sizeof(float), hipMemcpyDeviceToHost);

        export_binary_float(output_image_folder + "gather_" + std::to_string(modeling->nz) + "x" + std::to_string(spread) + "_shot_" + std::to_string(modeling->geometry->sInd[modeling->srcId]+1) + ".bin", gather, modeling->nz*modeling->geometry->spread[modeling->srcId]);
    }

    hipMemcpy(image, d_image, modeling->nPoints*sizeof(float), hipMemcpyDeviceToHost);

    # pragma omp parallel for
    for (int index = 0; index < modeling->nPoints; index++)
        image[index] *= 1.0f / modeling->geometry->nrel;
}

__global__ void cross_correlation(float * Ts, float * Tr, float * image, float * gather, float * seismic, float aperture, float cmp, int nPoints, int spread, int nz, int nt, float dt, float dx, float dz)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < nPoints)
    {
        int i = (int)(index % nz);
        int j = (int)(index / nz);

        float sigx = tanf(aperture*PI/180.0f)*i*dz;        
        float value = expf(-0.5*powf((j*dx - cmp)/(sigx + 1e-6f), 2.0f));

        float T = Ts[index] + Tr[index]; 
    
        int tId = (int)(T / dt);

        if (tId < nt) 
        {
            image[index] += value * seismic[tId + spread*nt];

            if (j == (int)(cmp/dx))
                gather[i + spread*nz] = value * seismic[tId + spread*nt];
        }    
    }
}
