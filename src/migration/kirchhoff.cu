#include "hip/hip_runtime.h"
# include "kirchhoff.cuh"

void Kirchhoff::set_specifications()
{
    hipMalloc((void**)&(d_Tr), modeling->nPoints*sizeof(float));
    hipMalloc((void**)&(d_Ts), modeling->nPoints*sizeof(float));
    hipMalloc((void**)&(d_image), modeling->nPoints*sizeof(float));

    hipMalloc((void**)&(d_seismic), modeling->nt*modeling->max_spread*sizeof(float));

    nThreads = 256;
    nBlocks = (int)((modeling->nPoints + nThreads - 1) / nThreads);
}

void Kirchhoff::run_cross_correlation()
{
    hipMemset(d_image, 0.0f, modeling->nPoints*sizeof(float));

    for (modeling->srcId = 0; modeling->srcId < modeling->geometry->nrel; modeling->srcId++)
    {
        read_seismic_data();

        modeling->show_information();

        std::cout << "\nKirchhoff depth migration: computing image matrix\n";

        modeling->initialization();
        modeling->forward_solver();

        modeling->reduce_boundary(modeling->T, Ts);

        hipMemcpy(d_Ts, Ts, modeling->nPoints*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_seismic, seismic, modeling->nt*modeling->geometry->spread[modeling->srcId]*sizeof(float), hipMemcpyHostToDevice);
        
        int spread = 0;

        float sx = modeling->geometry->xsrc[modeling->geometry->sInd[modeling->srcId]];

        for (modeling->recId = modeling->geometry->iRec[modeling->srcId]; modeling->recId < modeling->geometry->fRec[modeling->srcId]; modeling->recId++)
        {
            import_binary_float(output_table_folder + "traveltimes_receiver_" + std::to_string(modeling->recId+1) + ".bin", Tr, modeling->nPoints);

            float rx = modeling->geometry->xrec[modeling->recId];

            float cmp = sx + 0.5f*(rx - sx);

            hipMemcpy(d_Tr, Tr, modeling->nPoints*sizeof(float), hipMemcpyHostToDevice);

            cross_correlation<<<nBlocks, nThreads>>>(d_seismic, d_Ts, d_Tr, d_image, aperture, cmp, modeling->nPoints, spread, modeling->nz, modeling->nt, modeling->dt, modeling->dx, modeling->dz);

            ++spread;
        }
    }

    hipMemcpy(image, d_image, modeling->nPoints*sizeof(float), hipMemcpyDeviceToHost);

    # pragma omp parallel for
    for (int index = 0; index < modeling->nPoints; index++)
        image[index] *= 1.0f / modeling->geometry->nrel;
}

__global__ void cross_correlation(float * seismic, float * Ts, float * Tr, float * image, float aperture, float cmp, int nPoints, int spread, int nz, int nt, float dt, float dx, float dz)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < nPoints)
    {
        int i = (int)(index % nz);
        int j = (int)(index / nz);

        float sigx = tanf(aperture*PI/180.0f)*i*dz;        
        float value = expf(-0.5*powf((j*dx - cmp)/(sigx + 1e-6f), 2.0f));

        float T = Ts[index] + Tr[index]; 
    
        int tId = (int)(T / dt);

        if (tId < nt) image[index] += value * seismic[tId + spread*nt];
    }
}
