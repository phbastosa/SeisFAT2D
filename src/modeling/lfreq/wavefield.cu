#include "hip/hip_runtime.h"
# include "wavefield.cuh"

void Wavefield::set_specifications()
{
    nt = std::stoi(catch_parameter("time_samples", parameters));
    dt = std::stof(catch_parameter("time_spacing", parameters));

    fmax = std::stof(catch_parameter("max_frequency", parameters));

    set_wavelet();
    set_boundaries();
    set_properties();    
    set_conditions();    

    nThreads = 256;
    nBlocks = (int)(matsize / nThreads) + 1;

    current_xrec = new int[max_spread]();
    current_zrec = new int[max_spread]();

    define_cerjan_dampers();

    hipMalloc((void**)&(rIdx), max_spread*sizeof(int));
    hipMalloc((void**)&(rIdz), max_spread*sizeof(int));
}

void Wavefield::set_wavelet()
{
    float * signal_aux1 = new float[nt]();
    float * signal_aux2 = new float[nt]();

    float pi = 4.0f*atanf(1.0f);
    float t0 = 2.0f*sqrtf(pi) / fmax;
    float fc = fmax / (3.0f * sqrtf(pi));

    tlag = (int)(t0 / dt) + 1;

    for (int n = 0; n < nt; n++)
    {
        float td = n*dt - t0;

        float arg = pi*pi*pi*fc*fc*td*td;

        signal_aux1[n] = 1e5f*(1.0f - 2.0f*arg)*expf(-arg);
    }

    for (int n = 0; n < nt; n++)
    {
        float summation = 0;
        for (int i = 0; i < n; i++)
            summation += signal_aux1[i];    
        
        signal_aux2[n] = summation;
    }

    


    // export_binary_float("wavelet_original.bin", signal_aux2, nt);
    // export_binary_float("wavelet_modified.bin", signal_aux1, nt);



    // hipMalloc((void**)&(wavelet), nt*sizeof(float));

    // hipMemcpy(wavelet, signal, nt*sizeof(float), hipMemcpyHostToDevice);

    // delete[] aux_s;
    // delete[] signal;
}

void Wavefield::set_boundaries()
{
    nb = std::stoi(catch_parameter("boundary_samples", parameters));

    nxx = nx + 2*nb;
    nzz = nz + 2*nb;

    matsize = nxx*nzz;
}

void Wavefield::define_cerjan_dampers()
{
    float * damp1D = new float[nb]();
    float * damp2D = new float[nb*nb]();

    float factor = std::stof(catch_parameter("boundary_damper", parameters));

    for (int i = 0; i < nb; i++) 
    {
        damp1D[i] = expf(-powf(factor * (nb - i), 2.0f));
    }

    for(int i = 0; i < nb; i++) 
    {
        for (int j = 0; j < nb; j++)
        {   
            damp2D[j + i*nb] += damp1D[i]; // up to bottom
            damp2D[i + j*nb] += damp1D[i]; // left to right
        }
    }

    for (int index = 0; index < nb*nb; index++)
        damp2D[index] -= 1.0f;

	hipMalloc((void**)&(d1D), nb*sizeof(float));
	hipMalloc((void**)&(d2D), nb*nb*sizeof(float));

	hipMemcpy(d1D, damp1D, nb*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d2D, damp2D, nb*nb*sizeof(float), hipMemcpyHostToDevice);

    delete[] damp1D;
    delete[] damp2D;
}

